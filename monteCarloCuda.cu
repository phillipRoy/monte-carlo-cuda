
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <ctime>
#include <chrono>
#include <random>


#define N 100000000
#define TIMING
#ifdef TIMING
#define INIT_TIMER auto start = std::chrono::high_resolution_clock::now();
#define START_TIMER start = std::chrono::high_resolution_clock::now();
#define STOP_TIMER std::cout << "Runtime of " << N << ": " << \
	std::chrono::duration_cast<std::chrono::milliseconds>( \
		std::chrono::high_resolution_clock::now()-start \
	).count() << " ms " << std:: endl;
#else
#define INIT_TIMER
#define START_TIMER
#define START_TIMER()
#endif

__global__ void buildPoint(double *a, double *b, double *c) {
	if(pow(a[blockIdx.x], 2) + pow(b[blockIdx.x],2) <= 0.25) {
		c[blockIdx.x] = 1.0;
	}
}

int main(void) {
	INIT_TIMER
	START_TIMER
	std::random_device rd;
	std::mt19937 eng(rd());
	std::uniform_real_distribution<> distr(-0.5, 0.5);
	double *a, *b, *c;
	double *d_a, *d_b, *d_c;
	int size = N * sizeof(double);
	
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	
	a = (double *)malloc(size);
	b = (double *)malloc(size);
	c = (double *)malloc(size);
	
	for(int i = 0; i < N; ++i) {
		a[i] = distr(eng);
		b[i] = distr(eng);
	}
	
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	buildPoint<<<N,1>>>(d_a, d_b, d_c);
	
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	
	double pointsIn = 0;
	for(int i = 0; i < N; ++i)
		pointsIn += c[i];
		
	STOP_TIMER
	
	std::cout.precision(9);
	std::cout << (4 * pointsIn) / N << std::endl;
	
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}